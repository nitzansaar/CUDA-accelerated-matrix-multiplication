#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplyGPU(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024; // Default matrix size
    if (argc > 1) {
        N = atoi(argv[1]);
    }
    
    size_t size = N * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    
    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (float)(rand() % 100) / 10.0f;
        h_B[i] = (float)(rand() % 100) / 10.0f;
    }
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start time
    hipEventRecord(start);
    
    matrixMultiplyGPU<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Output timing information
    printf("%d,%.6f\n", N, milliseconds);
    
    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
} 